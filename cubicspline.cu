#include "hip/hip_runtime.h"
// -------------------------------------------------------------
// cuDPP -- CUDA Data Parallel Primitives library
// -------------------------------------------------------------
// $Revision$
// $Date$
// ------------------------------------------------------------- 
// This source code is distributed under the terms of license.txt in
// the root directory of this source distribution.
// ------------------------------------------------------------- 

/*
 * This is a basic example of how to use the CUDPP library.
 */

// includes, system
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <math.h>
#include <time.h>
#include <cstdlib>
#include <cstdio>

#define USE_CUSPARSE   true
// includes, project
#if defined(USE_CUDPP)
#include "cudpp.h"
#elif defined(USE_CUSPARSE)
#include "hipsparse.h"
#endif

#include <hip/hip_runtime_api.h>
#include "cuda_util.h"

#define CUDPP_APP_COMMON_IMPL
#include "common_config.h"
#include "stopwatch.h"

#include <string>


using namespace cudpp_app;

////////////////////////////////////////////////////////////////////////////////
// declaration, forward
void runTest( int argc, char** argv);
void cubicSpline();
void testing(int,int);
void testCases();

template <typename T> __global__ void _prepare_systems(T* a,T* b,T* c,T* d,const T* x,const T* y,T* h,const int n);
template <typename T> void preparing_parameters_gpu(T*,T*,T*,T*,const T*,T*,const T*,const int*,int,int,int);
template <typename T> void preparing_parameters_cpu(T*,T*,T*,T*,const T*,T*,const T*,const int*,int,int,int);

  
#define printArray(array,len) printArrayFmt(array,len,%f)


#define printArrayFmt(array,len,fmt) printf(#array"=["); \
    for(int i =0;i<len;i++){ \
        if(i % 10 == 0){ \
            printf("\n");   \
        }   \
        printf(#fmt", ",array[i]); \
    } \
    printf("]\n")
    
#define println(fmt,...) printf(fmt,##__VA_ARGS__);printf("\n")

#define blockalign(x,block_size) (x + block_size -1 )/ block_size

struct AddressPair_F
{
    float * host;
    float * device;
    /* data */
};

#if defined(USE_CUSPARSE)
void testCusparse(){
    hipsparseHandle_t handle=0;
    hipsparseStatus_t status;
    int version;

    status = hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS){
        println("CUSPARSE Library initializing failed.");
    }

    status = hipsparseGetVersion(handle,&version);
    if ( status != HIPSPARSE_STATUS_SUCCESS){
        println("CUSPRSE get version failed");
    }else{
        println("CUSPARSE Version:%d",version);
    }


}
#endif
////////////////////////////////////////////////////////////////////////////////
// Program main
////////////////////////////////////////////////////////////////////////////////
int
main( int argc, char** argv) 
{
    int deviceCount;
    hipGetDeviceCount(&deviceCount);
    if (deviceCount == 0) {
        fprintf(stderr, "error: no devices supporting CUDA.\n");
        exit(EXIT_FAILURE);
    }
    int dev = 0;
    if (argc > 1) {
        std::string arg = argv[1];
        size_t pos = arg.find("=");
        if (arg.find("device") && pos != std::string::npos) {
            dev = atoi(arg.c_str() + (pos + 1));
        }
    }
    if (dev < 0) dev = 0;
    if (dev > deviceCount-1) dev = deviceCount - 1;
    hipSetDevice(dev);

    // int numSMs;
    // hipDeviceGetAttribute(&numSMs, hipDeviceAttributeMultiprocessorCount, dev);

    hipDeviceProp_t prop;
    if (hipGetDeviceProperties(&prop, dev) == hipSuccess)
    {
        printf("Using device %d:\n", dev);
        printf("%s; global mem: %uB; compute v%d.%d; clock: %d kHz; multi processor count:%d\n",
               prop.name, (unsigned int)prop.totalGlobalMem, (int)prop.major, 
               (int)prop.minor, (int)prop.clockRate,prop.multiProcessorCount);
    }

#if defined(USE_CUSPARSE)
    testCusparse();
#endif

    // runTest( argc, argv);
    // cubicSpline();
    // testing(512,16);
    testCases();
}

//cuda kernel's thread and block can not expand without limit. So, can not only using num of block and thread 
    //to represent system scale. stride trick must be used.
    //this testing is used to check whether stride logic is correct.

    //by design, each block handle one interplation segment. block stride through all segments.
    //thread num of controled by the system size parameter. thread stride through all point inside a segment.

void testCases(){
    /* 
        
        Case 1. thread stride testing
    */
    testing(
        100000, //num of check points, determin the num of segments. < total blocks
        1024 //system size,  the size of a single system. determin the num of threads. < segment size(20)
    );
}

////////////////////////////////////////////////////////////////////////////////
//! Run a simple test for CUDA
////////////////////////////////////////////////////////////////////////////////
template <typename T>
__global__ void _prepare_systems(T* a,T* b,T* c,T* d,const T* x,const T* y,const int* check_point_idx,T* h,const int n)
{
    //blockDim == systemSize
    //gridDim = numOfSystem
    //blockIdx == systemIdx
    int ti_base = blockIdx.x * blockDim.x + threadIdx.x;
    // __shared__ T h[blockDim+2];

    //grid-stride
    for(int ti = ti_base;ti < n;ti += blockDim.x * gridDim.x){



    if (ti >= n)
        return;

    if(ti == 0){
        //head

        // h[1] = 0.0f;
        d[ti] = 6.0f*(y[check_point_idx[1]] - y[check_point_idx[0]])/(x[check_point_idx[1]] - x[check_point_idx[0]])/ (x[check_point_idx[1]] - x[check_point_idx[0]]);
        h[ti] = 0.0f;
    }else if (ti < n -1){
        //middle
        /* shared mem version
        if( threadIdx.x == 0){
            //head thread
            h[0] = ti > 2 ? x[ti-1] - x[ti -2]:0.0f;
        }else if(threadIdx.x == blockDim.x -1){
            //last thread
            h[threadIdx.x + 2] = x[ti +1 ] - x[ti];
        }
        */
        
        for(int j=0;j<3;j++){
            d[ti] += 6.0f * y[check_point_idx[ti-1 + j]] / (x[check_point_idx[ ti-1 + j]] - x[check_point_idx[ti-1 + (j+1)%3]]) / (x[check_point_idx[ ti-1+j]] - x[check_point_idx[ti-1 + (j+2)%3]]);
        }
        h[ti] = x[check_point_idx[ti]] - x[check_point_idx[ti-1]]; 
        
    }else{
        //last ti = n-1
        
        d[ti] = 6.0f * (0.0f - (y[check_point_idx[ ti]] - y[check_point_idx[ti-1]])/(x[check_point_idx[ti]] - x[check_point_idx[ti-1]]) ) / (x[check_point_idx[ti]] - x[check_point_idx[ti-1]]);
        h[ti] = x[check_point_idx[ ti]] - x[check_point_idx[ti-1]];
    }

    if(threadIdx.x == blockDim.x -1 && ti < n - 1){
        //block can not be synchronized, so calculate the h[ti + 1] for the last ti in block. do not exceed n -1.
        h[ti + 1]  = x[check_point_idx[ti + 1]] - x[check_point_idx[ti]];
    }
    
    __syncthreads();        
    b[ti] = 2.0f;
    a[ti] = ti < n - 1 ? h[ti] / (h[ti] + h[ti +1]) : 1.0f;
    // c[ti] = h[threadIdx.x+2] / (h[threadIdx.x+1] + h[threadIdx.x +2]);
    c[ti] = ti < n - 1 ? h[ti +1 ] / (h[ti+1] + h[ti]):0.0f;
    }
}

//deprecated
template <typename T>
__global__ void _cubic_spline_segment(const T* x,const T* y ,const T* m,const T* h,const T* spline_x,T* spline_y, int segment_len){
    int ti = blockIdx.x * blockDim.x + threadIdx.x;
    if ( ti >= segment_len)
        return;

    float temp1 = x[1] - spline_x[ti];
    float temp2 = spline_x[ti] - x[0];
    spline_y[ti] = m[0] * powf(temp1,3) / (6 * h[0])
    + m[1] * powf(temp2,3) / (6*h[0])
    + (y[0] - m[0]*powf(h[0],2)/6) * (temp1) / h[0]
    + (y[1] - m[1]*powf(h[0],2)/6) * (temp2) / h[0]
    ;
}

template <typename T>
__global__ void _cubic_spline(const T* x,T* y,const int* check_point_idx,const T* m,const T* h,int totalSize){
    //grid-stride
    for( int i = blockIdx.x; i < totalSize; i += gridDim.x){
        int to_idx = check_point_idx[i];
        int from_idx = 0;
        if ( i > 0){
            from_idx = check_point_idx[i-1];
        }
        //block-stride
        for( int j = from_idx + threadIdx.x + 1; j < to_idx; j += blockDim.x){
            y[j] = m[i-1] * powf(x[to_idx] - x[j],3) / (6*h[i])
                + m[i] * powf(x[j] - x[from_idx],3) / (6*h[i])
                + (y[from_idx] - m[i-1] * powf(h[i],2)/6) * (x[to_idx] - x[j]) / h[i]
                + (y[to_idx] - m[i] * powf(h[i],2)/6) * (x[j] - x[from_idx]) / h[i]
                ;
        }

    }
}
template <typename T>
__global__ void _cubic_spline2(const T* x,T* y,const int* check_point_idx,const T* m,const T* h,int totalSize)
{
    for ( int i = threadIdx.x; i < totalSize; i += blockDim.x)
    {
        int to_idx = check_point_idx[i];
        int from_idx = 0;
        if ( i > 0 ){
            from_idx = check_point_idx[i -1 ];
        }

        for( int j = from_idx + blockIdx.x + 1; j < to_idx; j+= gridDim.x)
        {
            y[j] = m[i-1] * powf(x[to_idx] - x[j],3) / (6*h[i])
                + m[i] * powf(x[j] - x[from_idx],3) / (6*h[i])
                + (y[from_idx] - m[i-1] * powf(h[i],2)/6) * (x[to_idx] - x[j]) / h[i]
                + (y[to_idx] - m[i] * powf(h[i],2)/6) * (x[j] - x[from_idx]) / h[i]
                ;   
        }
    }
}

template <typename T>
void preparing_parameters_gpu(
    T* d_a,
    T* d_b,
    T* d_c,
    T* d_d,
    const T* d_x,
    T* d_diff,
    const T* d_y,
    const int* d_check_point_idx,
    const int len,
    const int systemSize,
    const int numOfSystem
    )
{
    if(systemSize * numOfSystem < len){
        println("invalid systemSize:%d and numOfSystem:%d parameter, less than total size:%d",systemSize,numOfSystem,len);
        return;
    }
    
    _prepare_systems<T><<<28*4,1024>>>(d_a,d_b,d_c,d_d,d_x,d_y,d_check_point_idx,d_diff,len);
}

template <typename T>
void preparing_parameters_cpu(
    T* a,
    T* b,
    T* c,
    T* d,
    const T* x,
    T* diff,
    const T* y,
    const int * check_point_idx,
    const int len,
    const int systemSize,
    const int numOfSystem
    )
{
    diff[0] = 0.0f;
    for(int i =1;i<len;i++)
    {
        diff[i] = x[check_point_idx[i] ] - x[check_point_idx[i-1]];
    }

    for(int s=0;s<numOfSystem;s++)
    {
        int base = s*systemSize;
        float* sub_b = b + base;
        float* sub_a = a + base;
        float* sub_c = c + base;
        float* sub_d = d + base;
        float* sub_diff = diff + base;
        // const float* sub_x = x + base;
        const int* sub_x_idx = check_point_idx + base;
        const float* sub_y = y + base;

        int last = len -1;
        for(int i=0;i<systemSize && i+base < len;i++)
        {

            sub_b[i] = 2.0f;
            if( i + base ==0){
                sub_a[i] = 0.0f;
                sub_c[i] = 1.0f;
                sub_d[i] = 6.0f*(y[sub_x_idx[1]] - y[sub_x_idx[0]])/(x[sub_x_idx[1]] - x[sub_x_idx[0]])/ (x[sub_x_idx[1]] - x[sub_x_idx[0]]);
                // sub_d[i] = 0.0f;
            }else if (i + base < last){
                sub_a[i] = sub_diff[i] / (sub_diff[i] + sub_diff[i+1]);
                sub_c[i] = sub_diff[i+1] / (sub_diff[i] + sub_diff[i+1]);
                sub_d[i] = 0.0f;
                // println("=============================");
                for(int j=0;j<3;j++){
                    sub_d[i] += 6.0f * y[sub_x_idx[i-1 + j]] / (x[sub_x_idx[i-1 + j]] - x[sub_x_idx[i-1 + (j+1)%3]]) / (x[sub_x_idx[i-1+j]] - x[sub_x_idx[i-1 + (j+2)%3]]);
                    // printf("y[%d] / (x[%d] - x[%d]) / (x[%d] - x[%d])\n",i-1+j,i-1+j,i-1+(j+1)%3,i-1+j,i-1+(j+2)%3);
                }
                // printf("=============================\n");
                // println("=============================");

            }else{
                // println("last i=%d y[i]=%f y[i-1]=%f",i,y[i],y[i-1]);
                sub_d[i] = 6.0f * (0.0f - (y[sub_x_idx[i]] - y[sub_x_idx[i-1]])/(x[sub_x_idx[i]] - x[sub_x_idx[i-1]]) ) / (x[sub_x_idx[i]] - x[sub_x_idx[i-1]]);
                // sub_d[i] = 0.0f;
                // printf("last element:%d@%f\n",i,sub_d[i]);
                sub_a[i] = 1.0f;
                sub_c[i] = 0.0f;

            }

            if(sub_d[i] != sub_d[i]){
                println("d[i+base] = %f, i+base=%d,i=%d",sub_d[i],i+base,i);
            }
        }
    }

}

void testing(int len,int SYSTEM_SIZE){
    //GPU
    // int len = 64;
    // int SYSTEM_SIZE = 32; // one system per block, so system size should be equal to block size (num of threads per block).
    int numOfblocks = blockalign(len,SYSTEM_SIZE);
    int numOfSystem = max(len / SYSTEM_SIZE,1);
    int memSize = len * sizeof(float);
    int spline_len;
    int part_i;
    hipsparseHandle_t handle=0;
    hipsparseStatus_t status;

    float a_error= 0.0f;
    float b_error = 0.0f;
    float c_error= 0.0f;
    float d_error = 0.0f;
    float m_error = 0.0f;

    int segment_start = 1;
    int segment_end = 1;
    int offset = 0;
    int residual = 0;



    cudpp_app::StopWatch timer;
    


    float * d_a;
    float * d_b;
    float * d_c;
    float * d_d;
    // float * d_x;
    // float * d_y;
    int * d_check_point_idx;
    float * d_diff;
    float * d_spline_x;
    float * d_spline_y;
    float * d_m;

    /*
    Malloc memories
    */
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_a,memSize));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_b,memSize));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_c,memSize));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_d,memSize));
    // CUDA_SAFE_CALL( hipMalloc( (void**) &d_x,memSize));
    // CUDA_SAFE_CALL( hipMalloc( (void**) &d_y,memSize));
    // CUDA_SAFE_CALL( hipMalloc( (void**) &d_m,memSize));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_diff,memSize));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_check_point_idx,memSize) );


    CUDA_SAFE_CALL( hipMemset(d_a,0,memSize) );
    CUDA_SAFE_CALL( hipMemset(d_b,0,memSize) );
    CUDA_SAFE_CALL( hipMemset(d_c,0,memSize) );
    CUDA_SAFE_CALL( hipMemset(d_d,0,memSize) );
    CUDA_SAFE_CALL( hipMemset(d_diff,0,memSize) );

    
    int * x = (int*) malloc(len * sizeof(int));
    float * y = (float*) malloc(memSize);
    float * spline_x;
    float * spline_y;
/*
For Testing
*/
    float * a2 = (float*) malloc(memSize);
    float * b2 = (float*) malloc(memSize);
    float * c2 = (float*) malloc(memSize);
    float * d2 = (float*) malloc(memSize);
    float * h2 = (float*) malloc(memSize);
    float * m2 = (float*) malloc(memSize);

    float * a = (float*) malloc(memSize);
    float * b = (float*) malloc(memSize);
    float * c = (float*) malloc(memSize);
    float * d = (float*) malloc(memSize);
    float * h = (float*) malloc(memSize);
    float * m = (float*) malloc(memSize);

    

    AddressPair_F addressPairs[] = {
        {a2,d_a},
        {b2,d_b},
        {c2,d_c},
        {d2,d_d},
        {h2,d_diff}
    };

/*==========================
Fake data
===========================*/
    spline_len = (len -1) * 20 + 1;
    spline_x = (float*) malloc(spline_len * sizeof(float));
    spline_y = (float*) malloc(spline_len * sizeof(float));
    memset(spline_y,0,spline_len * sizeof(float));

    for(int i=0;i<spline_len;i++){
        spline_x[i] = ((float)i) * 0.05;
    }


    for(int i = 0;i<len;i++){
        x[i] = i * 20;
        spline_y[x[i]] = cos(spline_x[x[i]]);
    }

    

    

    CUDA_SAFE_CALL( hipMalloc( (void**) &d_spline_x,spline_len*sizeof(float)));
    CUDA_SAFE_CALL( hipMalloc( (void**) &d_spline_y,spline_len*sizeof(float)));


    CUDA_SAFE_CALL( hipMemcpy( d_spline_x, spline_x, spline_len * sizeof(float), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL( hipMemcpy( d_spline_y, spline_y, spline_len * sizeof(float), hipMemcpyHostToDevice));
    CUDA_SAFE_CALL( hipMemcpy( d_check_point_idx, x, memSize, hipMemcpyHostToDevice));


#if defined(USE_CUDPP)
    result = cudppCreate(&theCudpp);
    if(result != CUDPP_SUCCESS)
    {
        printf("Error initializing CUDPP Library.\n");
        goto end;
    }
#endif

    /* ================================
    GPU preparing parameters
    ==================================*/
    timer.reset();
    timer.start();
    preparing_parameters_gpu<float>(d_a,d_b,d_c,d_d,d_spline_x,d_diff,d_spline_y,d_check_point_idx,len,SYSTEM_SIZE,numOfblocks);
    hipDeviceSynchronize();
    timer.stop();
    println("preparing_parameters_gpu costs:%f",timer.getTime());

    

    

    for(int i =0;i<5;i++){
        AddressPair_F addr = addressPairs[i];
        hipError_t result = hipMemcpy(addr.host , addr.device, memSize, hipMemcpyDeviceToHost);
        if (result != hipSuccess) {
            printf("Error: %s\n", hipGetErrorString(result));
            goto end;
        }
    }

    /* ===============================
    CPU preparing parameters
    =================================*/
#if 1
    
    timer.reset();
    timer.start();
    preparing_parameters_cpu<float>(a,b,c,d,spline_x,h,spline_y,x,len,SYSTEM_SIZE,numOfblocks);
    
    timer.stop();
    println("preparing_parameters_cpu costs:%f",timer.getTime());
    

#endif
    
    /*
    printArray(a,len);
    printArray(a2,len);
    printArray(b,len);
    printArray(b2,len);
    printArray(c,len);
    printArray(c2,len);
    printArray(d,len);
    printArray(d2,len);
    */
    // printArray(y,len);

/*============================
    GPU Solve Tridiagonal Matrix
============================*/
#if defined(USE_CUSPARSE)
    status = hipsparseCreate(&handle);
    if (status != HIPSPARSE_STATUS_SUCCESS){
        println("CUSPARSE Library initializing failed.");
    }
    d_m = d_d;
    timer.reset();
    timer.start();
    status = cusparseSgtsv(
        handle,
        len,1,
        d_a,
        d_b,
        d_c,
        d_d,
        len
    );
    hipDeviceSynchronize();
    timer.stop();
    if( status != HIPSPARSE_STATUS_SUCCESS){
        println("solve tridiagonal system failed.%d",status);
    }
    println("solve tridiagonal system cost:%f ms",timer.getTime());
    CUDA_SAFE_CALL( hipMemcpy( m, d_m, memSize, hipMemcpyDeviceToHost));
    //printArray(m,len);
#endif

#if defined(USE_CUDPP)
    result = cudppPlan(theCudpp, &tridiagonalPlan, config, 0, 0, 0);
    if (CUDPP_SUCCESS != result)
    {
        printf("Error creating CUDPPPlan here\n");
        goto end;
    }
    println("system size:%d num of system:%d",min(SYSTEM_SIZE,len),numOfSystem);

    err = cudppTridiagonal(tridiagonalPlan, 
                               d_a, 
                               d_b, 
                               d_c, 
                               d_d, 
                               d_m, 
                               min(SYSTEM_SIZE,len), 
                               numOfSystem);

    hipDeviceSynchronize();
    if (err != CUDPP_SUCCESS) 
    {
        printf("Error running cudppTridiagonal\n");
        goto end;
    }
    
    if( numOfSystem * SYSTEM_SIZE < len){
        offset = numOfSystem * SYSTEM_SIZE;
        println("residual system size: %d",len - offset);
        err = cudppTridiagonal(tridiagonalPlan,
                d_a + offset,
                d_b + offset,
                d_c + offset,
                d_d + offset,
                d_m + offset,
                len - offset,
                1
            );
        hipDeviceSynchronize();
        if (err != CUDPP_SUCCESS) 
        {
            printf("Error running cudppTridiagonal\n");
            goto end;
        }
    }
    CUDA_SAFE_CALL( hipMemcpy(m2,d_m,memSize,hipMemcpyDeviceToHost) );

    // printArray(m2,len);
#endif
//Cubic Spline
#if 1
    part_i = 0;

    
    


    // CUDA_SAFE_CALL( hipMemcpy( d_spline_x, spline_x, spline_len * sizeof(float), hipMemcpyHostToDevice));

    
    segment_start = 1;
    segment_end =1;
    part_i = 0;
    timer.reset();
    timer.start();
    /* =============================
        plan 1, kernel only handle single segment, repeatly call kernel to finish all spline. Slow
        =============================== */
    // for(int i =0;i<spline_len;i++){
    //     // println("spline_x:%f, x:%f",spline_x[i],x[part_i]);
    //     if( spline_x[i] == x[part_i] || i == spline_len -1) {
    //         segment_end = i;
    //         // println("segment [%d,%d) - %d",segment_start,segment_end,part_i);
    //         if(segment_end - segment_start > 0 && part_i > 0){
    //             _cubic_spline_segment<<< blockalign(segment_end - segment_start,512),512 >>>(
    //             d_x + part_i - 1,
    //             d_y + part_i - 1,
    //             d_m + part_i - 1,
    //             d_diff + part_i,
    //             d_spline_x + segment_start,
    //             d_spline_y + segment_start,
    //             segment_end - segment_start
    //             );
                
    //         }
    //         part_i ++;
    //         segment_start = i;
            
    //     }
    // }
    /* ==============================
        Plan 2, block handle entire segment, stride tought all segments.
    =================================*/
    println("before spline");       
    // printArray(spline_y,spline_len);
    // printArrayFmt(x,len,%d);
    _cubic_spline2<<< 28,SYSTEM_SIZE>>>(d_spline_x,d_spline_y,d_check_point_idx,d_m,d_diff,len);
    hipDeviceSynchronize();
    timer.stop();
    println("gpu cubic spline costs: %f",timer.getTime());

    

    CUDA_SAFE_CALL( hipMemcpy( spline_y,d_spline_y, spline_len * sizeof(float), hipMemcpyDeviceToHost));
    // spline_y[spline_len-1] = y[len -1];
    // printArray(spline_y,spline_len);
/* =============================
CPU Cubic Spline Interplation
===============================*/
#if 0

    

    memset(spline_y,0,spline_len * sizeof(float));

    for(int i=0;i<spline_len;i++){
        spline_x[i] = ((float)i) * 0.05;
    }


    for(int i = 0;i<len;i++){
        x[i] = i * 20;
        spline_y[x[i]] = cos(spline_x[x[i]]);
    }


    printf("preparing data done.\n");
    residual = len - min(SYSTEM_SIZE,len) * numOfSystem;

    testTridiagonalDataType<float>(a,b,c,d,m,min(SYSTEM_SIZE,len),numOfSystem,config);
    if(residual > 0){
        offset = min(SYSTEM_SIZE,len) * numOfSystem;
        testTridiagonalDataType<float>(a+offset,b+offset,c+offset,d+offset,m+offset,residual,1,config);
    }

    // printArray(spline_y,spline_len);
    part_i = 0;
    timer.reset();
    timer.start();
    for(int i =0;i<spline_len;i++){
        float x_i = spline_x[i];
        if ( x_i >= spline_x[x[part_i]] ){
            part_i ++;
        }
        float x_part_i = spline_x[x[part_i]];
        float y_part_i = spline_y[x[part_i]];
        float x_part_i_1 = spline_x[x[part_i -1]];
        float y_part_i_1 = spline_y[x[part_i -1]];

        spline_y[i] = m[part_i-1] * powf(x_part_i - x_i,3) / (6*h[part_i]) 
        + m[part_i] * powf(x_i - x_part_i_1,3) / (6 *h[part_i]) 
        + (y_part_i_1  - m[part_i -1 ] * powf(h[part_i],2) / 6) * (x_part_i - x_i) / h[part_i] 
        + (y_part_i     - m[part_i]     * powf(h[part_i],2) / 6) * (x_i - x_part_i_1) / h[part_i]
        ;
        // printf("%f,",spline[i]);
        // println("i=%d,part_i=%d,x_i=%f",i,part_i,x_i);
    }
    timer.stop();
    println("cpu cubic spline costs: %f",timer.getTime());
#endif
    // printArray(m,len);
    // printArray(m2,len);

    for(int i =0;i<len;i++){
        a_error += fabs(a[i] - a2[i]);
        b_error += fabs(b[i] - b2[i]);
        c_error += fabs(c[i] - c2[i]);
        d_error += fabs(d[i] - d2[i]);
        m_error += fabs(m[i] - m2[i]);
    }
    
    println("errors: \na:%f \nb:%f\nc:%f\nd:%f\nm:%f",a_error,b_error,c_error,d_error,m_error);
    
    // printArray(spline_x,spline_len);
    // printArray(spline_y,spline_len);
    // printArrayFmt(x,len,%d);
#endif
end:
    /*
    Free Memories
    */
#ifdef USE_CUDPP 
    cudppDestroy(theCudpp);
#endif
    println("free host memories");
    free(a);
    free(b);
    free(c);
    free(d);
    free(h);
    free(m);

    free(a2);
    free(b2);
    free(c2);
    free(d2);
    free(h2);
    free(m2);

    free(x);
    free(y);
    free(spline_x);
    free(spline_y);
    println("free device memories");
    CUDA_SAFE_CALL(hipFree(d_a));
    CUDA_SAFE_CALL(hipFree(d_b));
    CUDA_SAFE_CALL(hipFree(d_c));
    CUDA_SAFE_CALL(hipFree(d_d));
    // CUDA_SAFE_CALL(hipFree(d_x));
    // CUDA_SAFE_CALL(hipFree(d_y));
    // CUDA_SAFE_CALL(hipFree(d_m));
    CUDA_SAFE_CALL(hipFree(d_diff));
    CUDA_SAFE_CALL(hipFree(d_spline_x));
    CUDA_SAFE_CALL(hipFree(d_spline_y));


}
